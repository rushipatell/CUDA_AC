#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_DEPRECATE
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include<hip/hip_runtime.h>

#define CUDA_WARN(XXX) \
    do {if (XXX != hipSuccess) printf("%s\n", hipGetErrorString(XXX));} while (0)

#define BLOCK_ROW 32
#define BLOCK_COL 32

__global__ void kernal_process_image(float* image,float* image_updated, int height, int width, int kernel)
{
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;



    if (row>=0 && col>=0 && row<height && col<width && image[row*width+col]==0)
    {
        float div=0, su=0, wei=0;
        int i, j;
        for(i=-1; i<=1; i++)
            for(j=-1; j<=1; j++)
                if(i + row >= 0 && i + row < height && j + col >= 0 && j + col < width)
                    div+=image[(row+i)*width+col+j];

        if(div>0.06)
        {
            int range = kernel/2;
            for(i=-range; i<=range; i++){
                for(j=-range;j<=range;j++){
                    if(row+i<0 || row+i>height || col+j<0 || col+j>width || image[(row+i)*width+col+j]==0){
                        continue;
                    }
                    wei += 1 / sqrt( (float) ((i*i) + (j*j)));
                    su += (image[(row+i)*width+col+j]/sqrt(  (float) ( (i*i) + (j*j) )      ));
                }
            }
        }
        image_updated[row*width+col]=(wei!=0)?su/wei:0;
        // printf("%d,", row*width+col);
    }
    return;
}

float* process_image(float* input_image, int height, int width)
{
    float* image;
    float* image_updated;
    clock_t start_mem,end_mem;

    start_mem = clock();
    hipMalloc(&image, height*width*sizeof(float));
    hipMalloc(&image_updated, height*width*sizeof(float));

    hipMemcpy(image, input_image, height*width*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(image_updated, input_image, height*width*sizeof(float), hipMemcpyHostToDevice);
    end_mem = clock();

    double walltime=(double)(((double)(end_mem-start_mem)* 1000)/(double)CLOCKS_PER_SEC);
    printf("Time to Allocate & Copy data from Host to Device memory is: %f miliseconds\n",walltime);
    
    dim3 dimBlock(BLOCK_ROW, BLOCK_COL);
    dim3 dimGrid((height-1)/dimBlock.x + 1, (width-1)/dimBlock.y + 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    kernal_process_image <<<dimBlock, dimGrid>>> (image,image_updated, height, width, 5);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsed_time=0;
    hipEventElapsedTime(&elapsed_time, start, stop);

    printf("Time to identify black pixels and replace those with weighted average for image size : %d x %d is: %f miliseconds\n",height,width,elapsed_time);

    start_mem = clock();
    hipMemcpy(input_image, image_updated, height*width*sizeof(float), hipMemcpyDeviceToHost);
    end_mem = clock();

    walltime=(double)(((double)(end_mem-start_mem)* 1000)/(double)CLOCKS_PER_SEC);
    printf("Time to Allocate & Copy data from Device to Host memory is: %f miliseconds\n",walltime);

    hipFree(image);
    hipFree(image_updated);
    return input_image;
}


int main(int argc, char **argv)
{
    int i, j, x;
	clock_t start,end;
	unsigned char byte[54];

    if(argc<3)
    {
        printf("Insufficient input Argument");
        return 1;
    }

	start = clock();

	FILE* fIn = fopen(argv[1], "rb");//Input File name
	FILE* fOut = fopen(argv[2], "wb");//Output File name


	if (fIn == NULL)											// check if the input file has not been opened succesfully.
	{
		printf("File does not exist.\n");
	}

	end = clock();
	double walltime=(double)(((double)(end-start)* 1000)/(double)CLOCKS_PER_SEC);
    printf("Time to open input & output image is: %f miliseconds\n",walltime);


    start = clock();
	for (i = 0; i < 54; i++)											//read the 54 byte header from fIn
	{
		byte[i] = getc(fIn);
	}

    unsigned int width = *(int*)&byte[18];
	unsigned int height = *(int*)&byte[22];
	fwrite(byte, sizeof(unsigned char), 54, fOut);					//write the header back


    unsigned char temp_trial[1024];
    for(i=0;i<1024;i++)
	{
		temp_trial[i] = getc(fIn);
	}
	fwrite(temp_trial, sizeof(unsigned char), 1024, fOut); 



	end = clock();
	walltime=(double)(((double)(end-start)* 1000)/(double)CLOCKS_PER_SEC);
    printf("Time to read & write header file for size : %d x %d is: %f miliseconds\n",height,width,walltime);

	printf("width: %d\n", width);
	printf("height: %d\n", height);

    int size = height*width;

	unsigned char* buffer = (unsigned char*)malloc(size * sizeof(unsigned char));
	unsigned char* out = (unsigned char*)malloc(size * sizeof(unsigned char));
	float* c = (float*)malloc(size * sizeof(float));

	start = clock();
	for (i = 0; i < height; i++)
	{
		for (j = 0; j < width; j++)
		{
			buffer[i * width + j] = getc(fIn);
		}
	}

    end = clock();
	walltime=(double)(((double)(end-start)* 1000)/(double)CLOCKS_PER_SEC);
    printf("Time to read image file into buffer for size : %d x %d is: %f miliseconds\n",height,width,walltime);


    start = clock();
	for (i = 0; i < size; i++)
	{
		c[i] = ((float)(buffer[i])) / (255.0f);
	}
	end = clock();
	walltime=(double)(((double)(end-start)* 1000)/(double)CLOCKS_PER_SEC);
    printf("Time to convert pixel values in 0-1 range for image size : %d x %d is: %f miliseconds\n",height,width,walltime);

   float* image;
    float* image_updated;
    clock_t start_mem,end_mem;

    start_mem = clock();
    hipMalloc(&image, height*width*sizeof(float));
    hipMalloc(&image_updated, height*width*sizeof(float));

    hipMemcpy(image, c, height*width*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(image_updated, c, height*width*sizeof(float), hipMemcpyHostToDevice);
    end_mem = clock();

    double walltime=(double)(((double)(end_mem-start_mem)* 1000)/(double)CLOCKS_PER_SEC);
    printf("Time to Allocate & Copy data from Host to Device memory is: %f miliseconds\n",walltime);
    
    dim3 dimBlock(BLOCK_ROW, BLOCK_COL);
    dim3 dimGrid((height-1)/dimBlock.x + 1, (width-1)/dimBlock.y + 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    kernal_process_image <<<dimBlock, dimGrid>>> (image,image_updated, height, width, 5);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float elapsed_time=0;
    hipEventElapsedTime(&elapsed_time, start, stop);

    printf("Time to identify black pixels and replace those with weighted average for image size : %d x %d is: %f miliseconds\n",height,width,elapsed_time);

    start_mem = clock();
    hipMemcpy(c, image_updated, height*width*sizeof(float), hipMemcpyDeviceToHost);
    end_mem = clock();

    walltime=(double)(((double)(end_mem-start_mem)* 1000)/(double)CLOCKS_PER_SEC);
    printf("Time to Allocate & Copy data from Device to Host memory is: %f miliseconds\n",walltime);

    hipFree(image);
    hipFree(image_updated);



	start = clock();
	for (i = 0; i < size; i++)
	{
		x = (int)(c[i]*255.0f);
		out[i] = (unsigned char)x;
	}

	fwrite(out, sizeof(unsigned char), size, fOut);           //write image data back to the file
    end = clock();
    walltime=(double)(((double)(end-start)* 1000)/(double)CLOCKS_PER_SEC);
    printf("Time to convert pixel values in range of 0-255 & write image in output for image size : %d x %d is: %f miliseconds\n",height,width,walltime);

	fclose(fIn);
	fclose(fOut);

    free(c);
    free(buffer);
    free(out);
	return 0;
}
